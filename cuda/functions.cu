#include "hip/hip_runtime.h"
#include "header.cuh"

/*****************************************************************************
* subroutine inidat - Initialize Array
*****************************************************************************/
void inidat(int nx, bool **u)
{
	int ix, iy;
	for (ix = 0; ix <= nx - 1; ix++)
	{
		for (iy = 0; iy <= nx - 1; iy++)
		{
			u[ix][iy] = (bool)(ix * (nx - ix - 1) * iy * (nx - iy - 1));
		}
	}
}

/**************************************************************************
* subroutine prtdat - Print the results
**************************************************************************/
void prtdat(int nx, bool** u, char *fnam)
{
	int ix, iy;
	FILE *fp;
	fp = fopen(fnam, "w");
	for (ix = 0; ix < nx; ix++)
	{
		for (iy = 0; iy < nx; iy++)
		{
			fprintf(fp, "%6.1f", u[ix][iy]);
			if (iy != nx - 1)
			{
				fprintf(fp, " ");
			}
			else
			{
				fprintf(fp, "\n");
			}
		}
	}
	fclose(fp);
}

// Create 2D array with sequential memory positions
bool** SeqAllocate(int size_of_matrix) {
	bool* sequence = (bool*) malloc(size_of_matrix*size_of_matrix*sizeof(bool));
	bool** matrix = (bool**) malloc(size_of_matrix*sizeof(bool *));
	int i;
	for (i = 0; i<size_of_matrix; i++)
		matrix[i] = &(sequence[i*size_of_matrix]);

	return matrix;
}

// Free 2D array with sequential memory positions
void SeqFree(bool** memory_ptr)
{
	free(memory_ptr[0]);
	free(memory_ptr);
}

// Assigh value to sequential 2D Array which is in GPU
__global__ void Assign(bool *d_sequence, bool** d_matrix )
{
	int i = threadIdx.x;
	d_matrix[i] = &(d_sequence[i* blockDim.x]);
}

// Create 2D array with sequential memory positions in GPU
bool** cudaSeqAllocate(int size_of_matrix) {
	
	bool* d_sequence;
	hipMalloc((void **)&d_sequence, size_of_matrix*size_of_matrix*sizeof(bool));

	bool** d_matrix;
	hipMalloc((void ***)&d_matrix, size_of_matrix*sizeof(bool *));

	Assign << <1 , size_of_matrix >> >(d_sequence, d_matrix);
	hipDeviceSynchronize();

	return d_matrix;
}

// Free Kernel
__global__ void cudaFreeKernel(bool** memory_ptr, bool** d_memorystart)
{
	*d_memorystart = memory_ptr[0];
}

// Free 2D array with sequential memory positions in GPU
void cudaSeqFree(bool** memory_ptr)
{
	bool** d_memorystart;
	hipMalloc((void ***)&d_memorystart, sizeof(bool*) );
	cudaFreeKernel << <1, 1 >> >(memory_ptr, d_memorystart);
	bool* memorystart;
	hipMemcpy(&memorystart, d_memorystart, sizeof(bool*), hipMemcpyDeviceToHost);
	hipFree(memorystart);
	hipFree(memory_ptr);
}